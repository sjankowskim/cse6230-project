#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../../utils.hpp"
#include <thrust/equal.h>
#include <thrust/execution_policy.h>

#define BLOCK_SIZE       (256)
#define NUM_TRIALS      (1000)
#define N             (100000)

/*-------------------------------*
 | CODE WRITTEN IN THIS SECITON  |
 | WAS DONE BY AN LLM!           |
 *-------------------------------*/

__global__ void gpt_findMinimum(const int* array, int size, int* result) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data to shared memory
    if (i < size) {
        sdata[tid] = array[i];
    } else {
        sdata[tid] = INT_MAX;
    }

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        atomicMin(result, sdata[0]);
    }
}

__global__ void copilot_findMinimum(int *array, int size, int *result) {
    extern __shared__ int shared[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    shared[threadIdx.x] = (tid < size) ? array[tid] : INT_MAX;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared[threadIdx.x] = min(shared[threadIdx.x], shared[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicMin(result, shared[0]);
    }
}

__global__ void gemini_find_min(int* A, int n, int* min_value) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int local_min = INT_MAX; // Initialize local minimum

  // Check for threads exceeding valid array bounds
  if (i < n) {
    local_min = min(local_min, A[i]);
  }

  // Use shared memory for efficient reduction within a block
  __shared__ int shared_min[BLOCK_SIZE];
  shared_min[threadIdx.x] = local_min;
  __syncthreads();

  // Reduce shared memory array to find the block minimum
  for (int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
    if (threadIdx.x < stride) {
      shared_min[threadIdx.x] = min(shared_min[threadIdx.x], shared_min[threadIdx.x + stride]);
    }
    __syncthreads();
  }

  // Thread with index 0 within the block updates the global minimum (atomicMin not required)
  if (threadIdx.x == 0) {
    *min_value = shared_min[0];
  }
}

__global__ void my_find_min(int* A, int n, int* min_value) {
    int min = INT_MAX;
    for (int i = 0; i < n; i++) {
        if (A[i] < min) {
            min = A[i];
        }
    }
    *min_value = min;
}

/*-------------------------------*
 |         END SECTION           |
 *-------------------------------*/

int main() {
    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    Timer<std::nano> timer;
    bool assertion = true;

    // TODO: Setup initial variables
    int* in;
    int* d_result;

    // TODO: hipMalloc as needed
    hipMalloc(&in, N * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    // TODO: Setup CUB stuff as needed
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Min(
        d_temp_storage, temp_storage_bytes, in, d_result, N);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    for (int i = 0; i < 5; i++) {
        std::chrono::duration<double, std::nano> sum(0);

        switch (i) {
            case LIBRARY:
                printf("Testing library call!\n");
                break;
            case GPT3:
                printf("Testing GPT-3.5!\n");
                break;
            case GPT4:
                printf("Testing GPT-4!\n");
                break;
            case COPILOT:
                printf("Testing Copilot!\n");
                break;
            case GEMINI:
                printf("Testing Gemini!\n");
                break;
        }

        for (int j = 0; j < NUM_TRIALS; j++) {

            // TODO: Setup initial variables and hipMemcpy as needed.
            int temp[N];
            srand(std::time(nullptr));
            for (int k = 0; k < N; k++) {
                temp[k] = rand();
            }
            hipMemcpy(in, temp, N * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(d_result, 0, sizeof(int));

            switch (i) {
                case LIBRARY:
                    timer.start();
                    hipcub::DeviceReduce::Min(
                        d_temp_storage, temp_storage_bytes, in, d_result, N);
                    hipDeviceSynchronize();
                    timer.stop();
                    break;
                case GPT3:
                    timer.start();
                    gpt_findMinimum<<<num_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(int)>>>(in, N, d_result);
                    hipDeviceSynchronize();
                    timer.stop();
                    break;
                case GPT4:
                    timer.start();
                    my_find_min<<<1, 1>>>(in, N, d_result);
                    hipDeviceSynchronize();
                    timer.stop();
                    break;
                case COPILOT:
                    timer.start();
                    copilot_findMinimum<<<num_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(int)>>>(in, N, d_result);
                    hipDeviceSynchronize();
                    timer.stop();
                    break;
                case GEMINI:
                    timer.start();
                    gemini_find_min<<<num_blocks, BLOCK_SIZE>>>(in, N, d_result);
                    hipDeviceSynchronize();
                    timer.stop();
                    break;
            }
            
            if (j != 0) {
                sum += timer.getElapsedTimeChrono();
            }

            // TODO: Verify results with library
            if (i != 0) {
                int *intended_result;
                hipMalloc(&intended_result, sizeof(int));
                hipcub::DeviceReduce::Min(
                        d_temp_storage, temp_storage_bytes, in, intended_result, N);
                assertion = thrust::equal(thrust::device, d_result, d_result + 1, intended_result);
                int res1;
                int res2;
                hipMemcpy(&res1, d_result, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&res2, intended_result, sizeof(int), hipMemcpyDeviceToHost);
                hipFree(intended_result);
                if (!assertion) {
                    printf("\tintended_result: %d, actual result: %d\n", res2, res1);
                    break;
                }
            }
        }

        if (!assertion) {
            printf("\tIncorrect output! Continuing...\n");
            continue;
        }
        printf("\ttotal avg time (nanoseconds): %f\n", sum / (NUM_TRIALS - 1));
    }

    // TODO: Free as needed
    hipFree(d_result);
    hipFree(in);
    hipFree(d_temp_storage);
}