#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../utils.hpp"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

/*-------------------------------*
 | CODE WRITTEN IN THIS SECITON  |
 | WAS DONE BY CHATGPT!          |
 *-------------------------------*/

#define BLOCK_SIZE 256

__global__
void gpt_countOnes(const int* array, int size, int* result) {
    __shared__ int partialCounts[256]; // Shared memory for storing partial counts
    int tid = threadIdx.x;
    int blockSize = blockDim.x;
    int chunkSize = (size + gridDim.x - 1) / gridDim.x; // Chunk size for each block
    int start = blockIdx.x * chunkSize;
    int end = min(start + chunkSize, size);

    // Count occurrences of 1 within the chunk
    int count = 0;
    for (int i = start + tid; i < end; i += blockSize) {
        if (array[i] == 1) {
            count++;
        }
    }

    // Store partial count in shared memory
    partialCounts[tid] = count;
    __syncthreads();

    // Perform block-level reduction using warp shuffle
    for (int offset = blockSize / 2; offset > 0; offset /= 2) {
        if (tid < offset) {
            partialCounts[tid] += partialCounts[tid + offset];
        }
        __syncthreads();
    }

    // Store block-level count in global memory
    if (tid == 0) {
        atomicAdd(result, partialCounts[0]);
    }
}

/*-------------------------------*
 |         END SECTION           |
 *-------------------------------*/

void print_int_array(int* arr, int size) {
    int* temp = (int *) malloc(size * sizeof(int));
    if (temp == 0) {
        printf("malloc failed, ruh roh!\n");
        return;
    }
    hipMemcpy(temp, arr, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("----------------------\n");
    for (int i = 0; i < size; i++) {
        printf("[%d]: %d\n", i, temp[i]);
    }

    free(temp);
}

int main() {
    double sum;
    int const NUM_TRIALS = 1000;
    const int N = 1000000;
    int *in;
    int result;

    Timer<std::nano> timer;
    uint64_t time_taken;

    hipMalloc(&in, N * sizeof(int));

    for (int i = 0; i < 5; i++) {
        for (int i = 0; i < NUM_TRIALS; i++) {
            result = 0;

            int temp[N];
            for (int j = 0; j < N; j++) {
                temp[j] = j % 2;
            }

            hipMemcpy(in, temp, N * sizeof(int), hipMemcpyHostToDevice);

            int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

            switch (type) {
                case 0:
                    int *d_result;
                    hipMalloc(&d_result, sizeof(int));
                    timer.start();
                    gpt_countOnes<<<numBlocks, BLOCK_SIZE>>>(in, N, d_result);
                    timer.stop();
                    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(d_result);
                    break;
                case 1:
                    timer.start();
                    result = thrust::count(thrust::device, in, in + N, 1);
                    timer.stop();
                    break;
                case 2:
                    // TODO: ChatGPT-4
                    break;
            }

            time_taken = timer.getElapsedTime();

            // printf("count: %d\n", result);

            hipFree(in);

            sum += time_taken;
            printf("time taken for trial %d (nanoseconds): %ld\n", i, time_taken);
        }
        
        printf("total avg time (nanoseconds): %f\n", sum / NUM_TRIALS);
    }

    
}